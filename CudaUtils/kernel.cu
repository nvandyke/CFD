#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
//#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <assert.h>

// The launch configurator returned block size
int blockSize;
// The actual grid size needed, based on input size
int gridSize;

hipError_t multiplyWithCuda(const double* A, const double* B, double* C, int numElements);
hipError_t divideWithCuda(const double* A, const double* B, double* C, int numElements);
hipError_t addWithCuda(const double* A, const double* B, double* C, int numElements);
hipError_t subtractWithCuda(const double* A, const double* B, double* C, int numElements);
hipError_t MATmultiplyWithCuda(const double* A, const double* B, double* C, int numRows, int numCols);

__global__ void vectorMultiply(const double* A, const double* B, double* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] * B[i];
    }
}

__global__ void vectorDivide(const double* A, const double* B, double* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;


    if (i < numElements) {
        C[i] = A[i] / B[i];
    }
}

__global__ void vectorAdd(const double* A, const double* B, double* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

__global__ void vectorSubtract(const double* A, const double* B, double* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] - B[i];
    }
}

__global__ void matrixMultiply(const double* A, const double* B, double* C, int N, int M, int N2, int M2) {
    
    int ROW = blockIdx.x * blockDim.x + threadIdx.x;
    int COL = blockIdx.y * blockDim.y + threadIdx.y;
    
    //printf("%i %i,%i %i,%i %i,%i %i\n", gridDim.x, gridDim.y, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    printf("%i %i\n", ROW, COL);
    double tmpSum = 0;

    if (ROW >= N || COL >= M2) {
        //printf("hmmm\n");
        return;
    }
    // each thread computes one element of the block sub-matrix
    for (int i = 0; i < N; i++) {
        tmpSum += A[ROW * M + i] * B[i * M2 + COL];
        //printf("%i %i\n", A[ROW * M + i], B[i * M + COL]);
    }
    C[ROW * M2 + COL] = tmpSum;


}

int notmain()
{
    const int arraySize = 5;
    const double a[arraySize] = { 1, 2, 3, 4, 5 };
    const double b[arraySize] = { 10, 20, 30, 40, 50 };
    double c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = multiplyWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} * {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);
    
    cudaStatus = divideWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "divideWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} / {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    cudaStatus = addWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    cudaStatus = subtractWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "subtractWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} - {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to multiply vectors in parallel.
hipError_t multiplyWithCuda(const double* A, const double* B, double* C, int numElements)
{
    double*dev_a = 0;
    double*dev_b = 0;
    double*dev_c = 0;
    hipError_t cudaStatus;
    

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with multiple grids/blocks.
    vectorMultiply<<<gridSize, blockSize >>>(dev_a, dev_b, dev_c, numElements);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return cudaStatus;
}


// Helper function for using CUDA to multiply vectors in parallel.
hipError_t divideWithCuda(const double* A, const double* B, double* C, int numElements) {
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectorDivide<<<1, numElements >>> (dev_a, dev_b, dev_c, numElements);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "divideWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(const double* A, const double* B, double* C, int numElements) {
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectorAdd<<<1, numElements >>> (dev_a, dev_b, dev_c, numElements);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}


// Helper function for using CUDA to subtract vectors in parallel.
hipError_t subtractWithCuda(const double* A, const double* B, double* C, int numElements) {
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectorSubtract<<<1, numElements >>> (dev_a, dev_b, dev_c, numElements);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "subtractWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}

void cudaCheck() {
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    fprintf(stdout, "using %i multiprocessors\n", properties.multiProcessorCount);
    fprintf(stdout, "max threads per processor: %i\n", properties.maxThreadsPerMultiProcessor);
    fprintf(stdout, "number of concurrent jobs %i\n", properties.multiProcessorCount * properties.maxThreadsPerMultiProcessor);

    



}


// Helper function for using CUDA to multiply vectors in parallel.
hipError_t MATmultiplyWithCuda(const double* A, const double* B, double* C, int numRows_a, int numCols_a, int numRows_b, int numCols_b) {
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;
    int numElements_a = numRows_a * numCols_a;
    int numElements_b = numRows_b * numCols_b;
    int numElements_c = numRows_a * numCols_b;
    assert(numRows_b == numCols_a);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements_c * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements_a * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements_b * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements_a * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements_b * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with multiple grids/blocks.
    matrixMultiply << <1, dim3(numRows_a, numCols_b) >> > (dev_a, dev_b, dev_c, numRows_a, numCols_a, numRows_b, numCols_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements_c * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}





extern "C" {
    void wrapper(double*a, double* b, double* c, int numElements) {
        hipError_t cudaStatus = multiplyWithCuda(a, b, c, numElements);
        assert(cudaStatus == hipSuccess);
    }
    void cudaStart(int numElements) {
        int minGridSize;    // The minimum grid size needed to achieve the
                            // maximum occupancy for a full device
                            // launch
        
        hipOccupancyMaxPotentialBlockSize(
            &minGridSize,
            &blockSize,
            (void*)vectorMultiply,
            0,
            numElements);

        // Round up according to array size
        gridSize = (numElements + blockSize - 1) / blockSize;
        fprintf(stdout, "mesh %i, grid %i, block %i\n", numElements, gridSize, blockSize);
        

        /*
        int d = 3, e = 2;
        int f = 2, g = 4;
        int tot = d * e;
        const double a[6] = { 1,1,2,2,3,3 };
        const double b[8] = { 1,1,1,1,2,2,2,2 };
        double c[12] = { 0 };

        hipError_t error = MATmultiplyWithCuda(a, b, c, d, e, f, g);
        
        assert(error == hipSuccess);

        fprintf(stdout, "\n\n%f %f %f\n%f %f %f\n%f %f %f\n", c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7], c[8]);
        */
    }
    void cudaEnd() {
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
        }
        fprintf(stdout, "Cuda Freed\n");
    }
}