﻿
#include "hip/hip_runtime.h"

//#include <helper_cuda.h>

#include <stdio.h>
#include <assert.h>

hipError_t multiplyWithCuda(const double* A, const double* B, double* C, int numElements);
hipError_t divideWithCuda(const double* A, const double* B, double* C, int numElements);
hipError_t addWithCuda(const double* A, const double* B, double* C, int numElements);
hipError_t subtractWithCuda(const double* A, const double* B, double* C, int numElements);

__global__ void vectorMultiply(const double* A, const double* B, double* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] * B[i];
    }
}

__global__ void vectorDivide(const double* A, const double* B, double* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;


    if (i < numElements) {
        C[i] = A[i] / B[i];
    }
}

__global__ void vectorAdd(const double* A, const double* B, double* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

__global__ void vectorSubtract(const double* A, const double* B, double* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] - B[i];
    }
}

int notmain()
{
    const int arraySize = 5;
    const double a[arraySize] = { 1, 2, 3, 4, 5 };
    const double b[arraySize] = { 10, 20, 30, 40, 50 };
    double c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = multiplyWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} * {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);
    
    cudaStatus = divideWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "divideWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} / {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    cudaStatus = addWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    cudaStatus = subtractWithCuda(a, b, c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "subtractWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} - {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to multiply vectors in parallel.
hipError_t multiplyWithCuda(const double* A, const double* B, double* C, int numElements)
{
    double*dev_a = 0;
    double*dev_b = 0;
    double*dev_c = 0;
    hipError_t cudaStatus;
    int blockSize;      // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the
                        // maximum occupancy for a full device
                        // launch
    int gridSize;       // The actual grid size needed, based on input
                        // size

    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        (void*)vectorMultiply,
        0,
        numElements);

    // Round up according to array size
    gridSize = (numElements + blockSize - 1) / blockSize;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with multiple grids/blocks.
    vectorMultiply<<<gridSize, blockSize >>>(dev_a, dev_b, dev_c, numElements);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return cudaStatus;
}


// Helper function for using CUDA to multiply vectors in parallel.
hipError_t divideWithCuda(const double* A, const double* B, double* C, int numElements) {
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectorDivide<<<1, numElements >>> (dev_a, dev_b, dev_c, numElements);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "divideWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(const double* A, const double* B, double* C, int numElements) {
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectorAdd<<<1, numElements >>> (dev_a, dev_b, dev_c, numElements);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}


// Helper function for using CUDA to subtract vectors in parallel.
hipError_t subtractWithCuda(const double* A, const double* B, double* C, int numElements) {
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numElements * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, A, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, B, numElements * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectorSubtract<<<1, numElements >>> (dev_a, dev_b, dev_c, numElements);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "subtractWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_c, numElements * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}

void cudaCheck() {
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    fprintf(stdout, "using %i multiprocessors\n", properties.multiProcessorCount);
    fprintf(stdout, "max threads per processor: %i\n", properties.maxThreadsPerMultiProcessor);
    fprintf(stdout, "number of concurrent jobs %i\n", properties.multiProcessorCount * properties.maxThreadsPerMultiProcessor);

    



}


extern "C" {
    void wrapper(double*a, double* b, double* c, int numElements) {
        hipError_t cudaStatus = multiplyWithCuda(a, b, c, numElements);
        assert(cudaStatus == hipSuccess);
    }
}